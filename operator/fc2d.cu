#include "hip/hip_runtime.h"
#include"fc2d.h"

__global__ void WUpdate(float *data, float *grad, int size, int RST, float learning_rate)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= size) return;
    data[idx] += grad[idx % RST] * learning_rate;
    __syncthreads();
}

__global__ void FileOnes(float *data, int size)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= size) return;
    data[idx] = 1.0f;
    __syncthreads();
}
__global__ void BiasForward(float *data, const float *bias, int size, int c)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= size) return;
    data[idx] += bias[idx % c];
    __syncthreads();
}

Fc2d::Fc2d(int k) : K_(k)
{
    alpha          = 1.0f;
    beta           = 0.0f;
    p_bias_        = nullptr;
    p_input_       = nullptr;
    p_output_      = nullptr;
    p_weights_     = nullptr;
    grads_bias_    = nullptr;
    grads_data_    = nullptr;
    grads_weights_ = nullptr;
    onevec         = nullptr;
}

Fc2d::~Fc2d()
{
    delete p_weights_;
    free(grads_weights_);
    free(grads_data_);
    free(grads_bias_);
    free(onevec);
}

void Fc2d::AddInput(ITensor *input)
{
    this->p_input_ = dynamic_cast<Tensor4d*>(input);
}

ITensor *Fc2d::LayerInit()
{
    length_ = p_input_->C() * p_input_->H() * p_input_->W();
    if(this->p_weights_ == nullptr)
    {
        this->p_weights_ = new Tensor4d(K_, p_input_->C(), p_input_->H(), p_input_->W());
        p_weights_->Randomize();
    }
    if(this->p_output_ == nullptr)
    {
        p_output_ = new Tensor4d(p_input_->N(), K_, 1, 1);
    }
    if(this->p_bias_ == nullptr)
    {
        p_bias_ = new Tensor4d(1, K_, 1, 1);
        p_bias_->Randomize();
    }
    checkCudaError(hipMalloc(&onevec, sizeof(float) * p_input_->N()));
    FileOnes<<<(p_input_->N() + 255)/256, 256>>>(onevec, p_input_->N());
    return p_output_;
}

void Fc2d::Forward(bool del)
{
    Tensor4d *out = p_output_;
    checkCudaError(hipblasSgemm(Session::instance().cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N,
                               K_, p_input_->N(), length_,
                               &alpha,
                               p_weights_->GpuPointer(), length_,
                               p_input_->GpuPointer(),   length_,
                               &beta,
                               out->GpuPointer(),        K_
    ));
    log_info("Forward");
    p_output_->PrintAll();
    log_info("bias");
    p_bias_->PrintAll();
    BiasForward<<<(p_input_->Size() + 255)/256, 256>>>(p_output_->GpuPointer(), p_bias_->GpuPointer(), p_input_->Size(), K_);
    log_info("Add bias");
    p_output_->PrintAll();
}

float *Fc2d::Backward(float *down_grads, bool del)
{
    if (grads_weights_ == nullptr && grads_data_ == nullptr)
    {
        checkCudaError(hipMalloc(&grads_weights_, sizeof(float) * p_weights_->Size()));
        checkCudaError(hipMalloc(&grads_data_,    sizeof(float) * p_input_->Size()));
        checkCudaError(hipMalloc(&grads_bias_,    sizeof(float) * p_bias_->Size()));
    }
    checkCudaError(hipblasSgemm(Session::instance().cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T,
                               length_, K_, p_input_->N(),
                               &alpha,
                               p_input_->GpuPointer(), length_, 
                               down_grads,             K_,
                               &beta,
                               grads_weights_,         length_
    ));
    checkCudaError(hipblasSgemv(Session::instance().cublas_handle(), HIPBLAS_OP_N,
                               K_, p_input_->N(), 
                               &alpha,
                               down_grads,  K_,
                               onevec,      1,
                               &alpha,
                               grads_bias_, 1
    ));
    float *a = (float*)malloc(sizeof(float) * 10);
    if(a != nullptr)
    {
        checkCudaError(hipMemcpy(a, grads_bias_, sizeof(float) * 10, hipMemcpyDeviceToHost));
        std::cout << "fc bias gradients\n";
        for(int i = 0; i < 10; i++)
             std::cout << a[i] << ' ';
        std::cout << "\n";
        free(a);
    }

    checkCudaError(hipblasSgemm(Session::instance().cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                               length_, p_input_->N(), K_,
                               &alpha,
                               p_weights_->GpuPointer(), length_,
                               down_grads,               K_,
                               &beta,
                               grads_data_,              length_
    ));
    
    //float *b = (float*)malloc(sizeof(float) * 100);
    //if(b != nullptr)
    //{
    //    checkCudaError(hipMemcpy(b, grads_data_, sizeof(float) * 100, hipMemcpyDeviceToHost));
    //    std::cout << "fc data gradients\n";
    //    for(int i = 0; i < 100; i++)
    //        std::cout << b[i] << ' ';
    //    std::cout << "\n";
    //    free(b);
    //}
    return grads_data_;
}

void Fc2d::UpdateWeights(float learning_rate)
{
     int size = p_weights_->Size();
     int N = p_weights_->N();
     WUpdate<<<(size + 255) / 256, 256>>>(p_weights_->GpuPointer(), grads_weights_, size, size / N, learning_rate);
}

void Fc2d::SetWeights(float data)
{
     p_weights_->SetValue(data);
}
