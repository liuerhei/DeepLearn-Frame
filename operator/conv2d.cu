#include "hip/hip_runtime.h"
#include "conv2d.h"


__global__ void DUpdate(float *data, float *grad, int size, int RST, float learn)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= size) return;
    data[idx] += grad[idx % RST] * learn;
    __syncthreads();
}

__global__ void AddBias(float *data, float *bias, int size, int k)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= size) return;
    data[idx] += bias[idx % k];
    __syncthreads();
}

Conv2d::Conv2d(int k, int s, int t, Padding_t mode)
    : K_(k), S_(s), T_(t), padding_mode_(mode)
{
    alpha         = 1.0f;
    beta          = 0.0f;
    p_input_      = nullptr;
    p_output_     = nullptr;
    p_filter_     = nullptr;
    grads_filter_ = nullptr;
    grads_data_   = nullptr;
    grads_bias_   = nullptr;
    bias_         = nullptr;
}

Conv2d::~Conv2d()
{
    checkCudnn(hipdnnDestroyConvolutionDescriptor(desc_));
    delete p_filter_;
    delete p_input_;
    delete p_output_;
    delete bias_;
    free(grads_data_);
    free(grads_filter_);
    free(grads_bias_);
    std::cout << "Conv layer delete\n";
    std::cout << "Conv2dLayer Delete\n";
}

void Conv2d::AddInput(ITensor *input)
{
    this->p_input_ = dynamic_cast<Tensor4d*>(input);
    //std::cout << "The input tensor is\n";
    //p_input_->PrintK(100);
}

ITensor *Conv2d::LayerInit()
{
    if (this->p_filter_ == nullptr)
    {
        this->p_filter_ = new Filter4d(K_, p_input_->C(), S_, T_);
        p_filter_->Randomize();
    }
    // Init the space and weights of filter.
    
    int h = p_input_->H();
    int w = p_input_->W();
    int n = p_input_->N();
    //std::cout << p_filter_->CpuPointer() << ' ' << p_filter_->GpuPointer() << "\n";
    filterStrideA_[0] = 1;
    filterStrideA_[1] = 1;
    dilationA_[0] = 1;
    dilationA_[1] = 1;
    if(padding_mode_ == valid)
    {
        padA_[0] = 0;
        padA_[1] = 0;
        H_out = (h - S_ + 1) / filterStrideA_[0] + (h - S_ + 1) % filterStrideA_[0];
        W_out = (w - T_ + 1) / filterStrideA_[0] + (h - T_ + 1) % filterStrideA_[1];
    }
    else{
        H_out = h / filterStrideA_[0] + h % filterStrideA_[0];
        W_out = w / filterStrideA_[1] + w % filterStrideA_[1];
    }
    checkCudnn(hipdnnCreateConvolutionDescriptor(&desc_));
    checkCudnn(hipdnnSetConvolutionNdDescriptor(
        desc_, 2, padA_, filterStrideA_, dilationA_,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT
    ));
    C_out = K_;
    N_out = n;
    // compute the size of output
    
    if (this->p_output_ == nullptr)
    {
        p_output_ = new Tensor4d(N_out, C_out, H_out, W_out);
    }

    Tensor4d *out = dynamic_cast<Tensor4d*>(p_output_);
    checkCudnn(hipdnnGetConvolutionForwardAlgorithm(
        Session::instance().cudnn_handle(), p_input_->Desc(), p_filter_->Desc(), desc_,
        out->Desc(), HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo_
    ));
    checkCudnn(hipdnnGetConvolutionForwardWorkspaceSize(
        Session::instance().cudnn_handle(), p_input_->Desc(), p_filter_->Desc(), desc_,
        out->Desc(), algo_, &size_in_bytes
    ));
    // compute the filter backward workspace size
    size_t fsize_bytes, dsize_bytes;
    checkCudnn(hipdnnGetConvolutionBackwardFilterAlgorithm(
        Session::instance().cudnn_handle(), p_input_->Desc(), p_output_->Desc(), desc_,
        p_filter_->Desc(), HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &falgo_
    ));
    checkCudnn(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
        Session::instance().cudnn_handle(), p_input_->Desc(), p_output_->Desc(), desc_,
        p_filter_->Desc(), falgo_, &fsize_bytes
    ));
    // compute the data backward workspace size
    checkCudnn(hipdnnGetConvolutionBackwardDataAlgorithm(
        Session::instance().cudnn_handle(), p_filter_->Desc(), p_output_->Desc(), desc_,
        p_input_->Desc(), HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &dalgo_
    ));
    checkCudnn(hipdnnGetConvolutionBackwardDataWorkspaceSize(
        Session::instance().cudnn_handle(), p_filter_->Desc(), p_output_->Desc(), desc_,
        p_input_->Desc(), dalgo_, &dsize_bytes
    ));
    size_in_bytes = (size_in_bytes > fsize_bytes) ? size_in_bytes : fsize_bytes;
    size_in_bytes = (size_in_bytes > dsize_bytes) ? size_in_bytes : dsize_bytes;

    Session::instance().update_workspace_size(size_in_bytes);
    // compute the workspace size of convolution forward and backward

    if (this->bias_ == nullptr)
    {
        bias_ = new Tensor4d(1, C_out, H_out, W_out);
        // in lenet, the bias tensor shape is 1, channel, 1, 1
        bias_->Randomize(0.1);
    }
    p_output_->PrintShape();
    return p_output_;
}

void Conv2d::Forward(bool del)
{
    Tensor4d *out = dynamic_cast<Tensor4d*>(p_output_);
    checkCudnn(hipdnnConvolutionForward(
        Session::instance().cudnn_handle(), &alpha, p_input_->Desc(), p_input_->GpuPointer(),
        p_filter_->Desc(), p_filter_->GpuPointer(), desc_, algo_, 
        Session::instance().workspace(), Session::instance().workspace_size(),
        &beta, out->Desc(), out->GpuPointer() 
    ));
    //std::cout << "Conv layer input****************************\n";
    //p_input_->PrintK(100);
    //std::cout << "Conv layer output****************************\n";
    //out->PrintK(100);
    //std::cout << "conv layer bias******************************\n";
    //bias_->PrintK(100);
    //AddBias<<<(out->Size() + 255) / 256, 256>>>(out->GpuPointer(), bias_->GpuPointer(), out->Size(), bias_->Size());
    //checkCudnn(hipdnnAddTensor(
    //    Session::instance().cudnn_handle(), &alpha, bias_->Desc(), bias_->GpuPointer(), 
    //    &beta, out->Desc(), out->GpuPointer()
    //));
    //std::cout << "Conv layer add bias & out ****************************\n";
    //out->PrintK(100);
}

float *Conv2d::Backward(float *down_grads, bool del)
{
     if (grads_filter_ == nullptr && grads_data_ == nullptr)
     {
        checkCudaError(hipMalloc(&grads_filter_, sizeof(float) * p_filter_->Size()));
        checkCudaError(hipMalloc(&grads_data_,   sizeof(float) * p_input_->Size()));
        checkCudaError(hipMalloc(&grads_bias_,   sizeof(float) * bias_->Size()));
     }
     // TODO
     // Here maybe have BUG
     // Because the size of each layer make sence, so the space can allocate once.
     //checkCudnn(hipdnnConvolutionBackwardBias(
     //     Session::instance().cudnn_handle(), &alpha, p_output_->Desc(),
     //     down_grads, &beta, bias_->Desc(), grads_bias_
     //));
     checkCudnn(hipdnnConvolutionBackwardFilter(
          Session::instance().cudnn_handle(), &alpha, p_input_->Desc(), p_input_->GpuPointer(),
          p_output_->Desc(), down_grads, desc_, HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
          Session::instance().workspace(), Session::instance().workspace_size(),
          &beta, p_filter_->Desc(), grads_filter_
     ));
     checkCudnn(hipdnnConvolutionBackwardData(
          Session::instance().cudnn_handle(), &alpha, p_filter_->Desc(), p_filter_->GpuPointer(),
          p_output_->Desc(), down_grads, desc_, HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
          Session::instance().workspace(), Session::instance().workspace_size(),
          &beta, p_input_->Desc(), grads_data_
     ));

     //float *a = (float *)malloc(sizeof(float) * p_filter_->Size());
     //checkCudaError(hipMemcpy(a, grads_filter_, sizeof(float) * p_filter_->Size(), hipMemcpyDeviceToHost));
     //std::cout << "conv filter gradients\n";
     //for(int i = 0; i < p_filter_->Size(); ++i)
     //   std::cout << a[i] << ' ';
     ////   a[i] = i;
     ////checkCudaError(hipMemcpy(grads_filter_, a, sizeof(float) * p_filter_->Size(), hipMemcpyHostToDevice));
     //// This is a test, it seems that the gradients always is 0
     //std::cout << "\n";

     //float *b = (float *)malloc(sizeof(float) * p_input_->Size());
     //checkCudaError(hipMemcpy(b, grads_data_,  sizeof(float) * p_input_->Size(),   hipMemcpyDeviceToHost));
     //std::cout << "conv data gradients\n";
     //for(int i = 0; i < p_input_->Size(); ++i)
     //   std::cout << b[i] << ' ';
     //std::cout << "\n";
     //free(b);
     return grads_data_;
}

void Conv2d::UpdateWeights(float learning_rate)
{
    int size = p_filter_->Size();
    int K = p_filter_->K();
    //std::cout << "**************************************\n";
    //p_filter_->PrintK(10);
    DUpdate<<<(size + 255) / 256, 256>>>(p_filter_->GpuPointer(), grads_filter_, size, size / K, learning_rate);
    //DUpdate<<<(bias_->Size() + 255) / 256, 256>>>(bias_->GpuPointer(), grads_bias_, bias_->Size(), bias_->Size(), 1);
    //p_filter_->PrintK(10);
    //std::cout << "**************************************\n";
    //bias_->PrintK(100);
}

void Conv2d::SetWeights(float data)
{
    //p_filter_->SetValue(data);
    // p_filter_->randomize();
}

void Conv2d::ToFile(const char *fileprefix)
{
    std::stringstream ssf;
    ssf << fileprefix << ".bin";

    FILE *fp = fopen(ssf.str().c_str(), "w+");
    if(!fp)
    {
        log_error("FILE cannot open");
        exit(0);
    }
    //this->p_filter_->SyncToCpu();
    fwrite(this->p_filter_->CpuPointer(), sizeof(float), this->p_filter_->Size(), fp);
    fclose(fp);
}
