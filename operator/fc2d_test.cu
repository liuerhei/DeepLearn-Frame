#include "hip/hip_runtime.h"
#include"fc2d_test.h"

__global__ void WUpdate(float *data, float *grad, int size, int RST)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= size) return;
    data[idx] = data[idx] + grad[idx % RST] * 0.0001;
    __syncthreads();
}

Fc2d_test::Fc2d_test(int k) : K_(k)
{
    alpha          = 1.0f;
    beta           = 0.0f;
    p_input_       = nullptr;
    p_output_      = nullptr;
    p_weights_     = nullptr;
    grads_data_    = nullptr;
    grads_weights_ = nullptr;
}

Fc2d_test::~Fc2d_test()
{
    checkCudaError(hipblasDestroy(cublasHandle_));
    delete p_weights_;
    free(grads_weights_);
    free(grads_data_);
}

void Fc2d_test::AddInput(ITensor *input)
{
    this->p_input_ = dynamic_cast<Tensor4d*>(input);
}

ITensor *Fc2d_test::LayerInit()
{
    length_ = p_input_->C() * p_input_->H() * p_input_->W();
    if(this->p_weights_ == nullptr)
    {
        this->p_weights_ = new Tensor4d(K_, p_input_->C(), p_input_->H(), p_input_->W());
        p_weights_->Randomize();
    }
    // Create cublas Handle
    checkCudaError(hipblasCreate(&cublasHandle_));
    if(this->p_output_ == nullptr)
    {
        p_output_ = new Tensor4d(p_input_->N(), K_, 1, 1);
    }
    //std::cout << "Input Tensor shape: ";
    //p_input_->PrintShape();
    //std::cout << "Weights shape: ";
    //p_weights_->PrintShape();
    //std::cout << "Output Tensor shape: ";
    //p_output_->PrintShape();
    return p_output_;
}

void Fc2d_test::Forward(bool del)
{
    Tensor4d *out = p_output_;
    checkCudaError(hipblasSgemm(cublasHandle_, HIPBLAS_OP_T, HIPBLAS_OP_N,
                               K_, p_input_->N(), length_,
                               &alpha,
                               p_weights_->GpuPointer(), length_,
                               p_input_->GpuPointer(),   length_,
                               &beta,
                               out->GpuPointer(),        K_
    ));
    //std::cout << "Fc layer input********************\n";
    //p_input_->PrintK(100);
    //std::cout << "Fc layer output********************\n";
    //out->PrintK(100);
}

float *Fc2d_test::Backward(float *down_grads, bool del)
{
    if (grads_weights_ == nullptr && grads_data_ == nullptr)
    {
        checkCudaError(hipMalloc(&grads_weights_, sizeof(float) * p_weights_->Size()));
        checkCudaError(hipMalloc(&grads_data_,    sizeof(float) * p_input_->Size()));
    }
    checkCudaError(hipblasSgemm(cublasHandle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
                               length_, K_, p_input_->N(),
                               &alpha,
                               p_input_->GpuPointer(), length_, 
                               down_grads,             K_,
                               &beta,
                               grads_weights_,         length_
    ));

    //float *a = (float*)malloc(sizeof(float) * 1000);
    //if(a != nullptr)
    //{
    //    checkCudaError(hipMemcpy(a, grads_weights_, sizeof(float) * 1000, hipMemcpyDeviceToHost));
    //    std::cout << "fc weights gradients\n";
    //    for(int i = 0; i < 1000; i++)
    //        std::cout << a[i] << ' ';
    //    std::cout << "\n";
    //    free(a);
    //}

    checkCudaError(hipblasSgemm(cublasHandle_, HIPBLAS_OP_N, HIPBLAS_OP_N,
                               length_, p_input_->N(), K_,
                               &alpha,
                               p_weights_->GpuPointer(), length_,
                               down_grads,               K_,
                               &beta,
                               grads_data_,              length_
    ));
    
    //float *b = (float*)malloc(sizeof(float) * 100);
    //if(b != nullptr)
    //{
    //    checkCudaError(hipMemcpy(b, grads_data_, sizeof(float) * 100, hipMemcpyDeviceToHost));
    //    std::cout << "fc data gradients\n";
    //    for(int i = 0; i < 100; i++)
    //        std::cout << b[i] << ' ';
    //    std::cout << "\n";
    //    free(b);
    //}
    return grads_data_;
}

void Fc2d_test::UpdateWeights()
{
     //p_weights_->PrintK(100);
     int size = p_weights_->Size();
     int K = p_weights_->N();
     WUpdate<<<(size + 255) / 256, 256>>>(p_weights_->GpuPointer(), grads_weights_, size, size / K);
     //p_weights_->PrintK(100);
     
}

void Fc2d_test::SetWeights(float data)
{
     p_weights_->SetValue(data);
}
